
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void HelloWorldCuk() {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid == 0) {
        printf("Hello World on device!\n");
    }
}

int main(int argc, const char** argv) {
    // Invoking the kernel function
    HelloWorldCuk<<<1, 1>>>();

    hipDeviceSynchronize();

    return 0;
}