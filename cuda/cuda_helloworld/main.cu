
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_vector_types.h"

__global__ void HelloWorldCuk() {
    printf("Hello World on device!\n");
}

int main(int argc, const char** argv) {
    // Invoking the kernel function
    HelloWorldCuk<<<1, 1>>>();

    hipDeviceSynchronize();
    
    return 0;
}