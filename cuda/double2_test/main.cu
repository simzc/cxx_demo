
#include <hip/hip_runtime.h>
#include <iostream>
#include <array>
#include <vector>
#include "hip/hip_vector_types.h"

struct MyStruct {
    std::vector<std::array<double, 2>> array_a;
};

int main(int argc, const char** argv) {
    std::vector<double2> test_a;
    test_a.resize(5);
    for (auto& var : test_a) {
        var = make_double2(1.0, 2.0);
    }

    std::vector<double2> test_b;
    test_b = test_a;

    for (auto& var : test_b) {
        std::cout << "var.x: " << var.x << "\t"
                  << "var.y: " << var.y << std::endl;
    }

    // std::vector<std::array<double, 2>> array_a = {{1., 2.}, {3., 4.}};
    std::vector<std::array<double, 2>> array_a;
    // std::vector<std::array<double, 2>> array_b;

    // array_b = array_a;
    // for (auto& var : array_b) {
    //     std::cout << "array[0]: " << var.at(0) << "\t"
    //               << "array[1]: " << var.at(1) << std::endl;
    // }

    MyStruct struct_a;
    struct_a.array_a = array_a;

    MyStruct struct_b;
    struct_b = struct_a;

    return 0;
}